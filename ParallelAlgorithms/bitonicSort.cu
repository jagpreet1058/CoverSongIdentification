
#include <hip/hip_runtime.h>
//x: Pointer to matrix
//N: Size of each row
//NPow2: Size of each row rounded up to nearest power of 2
__global__ void bitonicSort(float* X, int N, int NPow2)
{
    extern __shared__ float x[];
    int N2 = NPow2 >> 1;
    int offset = blockIdx.x*N;
    int k = 0;
    int i, i1, i2;
    float min, max;
    int size = 2;
    int stride;
    //Step 0: Figure out K (number of batches per block)
    int K = N2 >> 9;
    if (K == 0) {
        K = 1;
    }

    //Step 1: Copy row corresponding to this block into shared memory
    //bearing in mind that for bitonic sort there are half
    //as many threads as there are numbers in each row
    for (k = 0; k < (K << 1); k++) {
        i1 = k*N2 + threadIdx.x;
        if (i1 < N) {
            x[i1] = (float)X[offset + i1];
        }
        else if (i1 < NPow2) {
            //NOTE: Assuming all numbers are nonnegative
            //so these dummy padding values will go first
            x[i1] = -1;
        }
    }
    __syncthreads();

    //Step 2: Perform bitonic sort
    while (size < NPow2 << 1) {
        stride = size >> 1;
        while (stride > 0) {
            for (k = 0; k < K; k++) {
                i = k*N2 + threadIdx.x;
                i1 = stride*2*(i/stride) + i%stride;
                i2 = i1 + stride;
                if (x[i1] < x[i2]) {
                    min = x[i1];
                    max = x[i2];
                }
                else {
                    min = x[i2];
                    max = x[i1];
                }
                if (i/(size/2)%2 > 0) {
                    x[i1] = min;
                    x[i2] = max;
                }
                else {
                    x[i1] = max;
                    x[i2] = min;
                }
            }
            stride = stride >> 1;
            __syncthreads();
        }
        size = size << 1;
    }

    //Step 3: Copy Result Back
    for (k = 0; k < (K << 1); k++) {
        i1 = k*N2 + threadIdx.x;
        if (i1 >= N) {
            break;
        }
        X[offset + (N-i1)] = x[i1];
    }
    __syncthreads();
}


__global__ void memtest(float* X, int N, int N2, int K)
{
    extern __shared__ float x[];
    int offset = blockIdx.x*N;
    int t = threadIdx.x;

    if (t < N) {
        X[offset+t] = (float)(blockIdx.x + t);
    }

    //Step 3: Copy Result Back
    /*for (k = 0; k < 2*K; k++) {
        i1 = t*2*K+k;
        if (i1 >= N) {
            break;
        }
        X[t] = t*k;
    }*/

}
